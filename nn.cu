#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>

#define INPUT_SIZE 784
#define HIDDEN_SIZE 1024
#define OUTPUT_SIZE 10
#define BATCH_SIZE 64
#define BLOCK_SIZE 16

typedef struct {
    float *weights1;
    float *weights2;
    float *weights3;  

    float *biases1;
    float *biases2;
    float *biases3;   

    float *grad_layer1;
    float *grad_layer2;
    float *grad_layer3;  
} NeuralNetwork;

typedef struct {
  float *x1;
  float *a1;

  float *x2;
  float *a2;
  
  float *x3;
  float *a3;

  float *losses;
} Outputs;

__global__ void linear_forward(int batch_size, int n, int out_w, float* input, float* weights, float* biases, float* output)
{
  const uint column = blockIdx.x*blockDim.x + threadIdx.x;
  const uint row = blockIdx.y*blockDim.y + threadIdx.y;
  if (row < batch_size && column < out_w)
  {
    output[row*out_w+column] = biases[column];
    for(int i = 0; i < n; i++)
    {
      output[row*out_w+column] += weights[i*out_w + column] * input[row*n + i];
    }
  }
}

__global__ void linear_backward(int batch_size, int n, int out_w, float* weights, float* biases, float* d_l, float* out_d_l)
{
  const uint column = blockIdx.x*blockDim.x + threadIdx.x;
  const uint row = blockIdx.y*blockDim.y + threadIdx.y;
  if (row < batch_size && column < out_w)
  {
    float dl = 0.f;
    for(int i = 0; i < n; i++)
    {
      float w = weights[i*out_w + column];
      dl += w*d_l[row*n + i];
    }
    out_d_l[row*out_w + column] = dl;
  }
}

__global__ void relu_forward(int w, int h, float* a, float* b)
{
  const uint column = blockIdx.x*blockDim.x + threadIdx.x;
  const uint row = blockIdx.y*blockDim.y + threadIdx.y;
  if (row < h && column < w)
  {
    float activation = a[row*w+column];
    b[row*w+column] =  activation > 0.f ? activation : 0.f;
  }
}

__global__ void relu_backwards(int w, int h, float* a, float* d_l, float* b)
{
  const uint column = blockIdx.x*blockDim.x + threadIdx.x;
  const uint row = blockIdx.y*blockDim.y + threadIdx.y;
  if (row < h && column < w)
  {
    float activation = a[row*w+column];
    b[row*w+column] = activation > 0.f ? d_l[row*w+column] : 0.f;
  }
}

__global__ void softmax(int w, int h, float* a, float* b)
{
  const uint col = blockIdx.x*blockDim.x + threadIdx.x;
  const uint row = blockIdx.y*blockDim.y + threadIdx.y;
  if (row < h && col < w)
  {
    //subtract with maxval for numeric stability
    float maxval = a[row*w];
    for (int i = 1; i<w; i++)
    {
      maxval = max(maxval, a[row*w + i]);
    }
    float divisor = 0.f;
    for (int i = 0; i<w; i++)
    {
      divisor += exp(a[row*w + i] - maxval);
    }
    b[row*w + col] = exp(a[row*w + col]-maxval)/(divisor);
  }
}

__global__ void init_kaiming_normal(int W, int H, float* matrix){
    const uint row = blockDim.x * blockIdx.x + threadIdx.x;
    const uint col = blockDim.y * blockIdx.y + threadIdx.y;

    if (row < H && col < W){
    hiprandState state;
    hiprand_init(42, row*W+col, 0, &state);
    matrix[row*W + col] = hiprand_normal(&state)*sqrtf(2.f/H);
    }
}

void init_parameters(float* weights, float* biases, int W, int H)
{
// weights
  dim3 numBlocks = dim3(ceil(W/(float)BLOCK_SIZE), ceil(H/(float)BLOCK_SIZE), 1);
  dim3 numThreadsPerBlock = dim3(BLOCK_SIZE, BLOCK_SIZE, 1);
  init_kaiming_normal<<<numBlocks, numThreadsPerBlock>>>(W, H, weights);

// biases
  numBlocks = dim3(ceil(H/(float)BLOCK_SIZE), 1, 1);
  numThreadsPerBlock = dim3(BLOCK_SIZE, 1, 1);
  init_kaiming_normal<<<numBlocks, numThreadsPerBlock>>>(1, H, biases);
}

void initialize_nn(NeuralNetwork* nn){
    hipMalloc(&nn->weights1, HIDDEN_SIZE * INPUT_SIZE * sizeof(float));
    hipMalloc(&nn->biases1, HIDDEN_SIZE * sizeof(float));
    hipMalloc(&nn->grad_layer1, HIDDEN_SIZE * BATCH_SIZE * sizeof(float));
    init_parameters(nn->weights1, nn->biases1, HIDDEN_SIZE, INPUT_SIZE);

    hipMalloc(&nn->weights2, HIDDEN_SIZE * HIDDEN_SIZE * sizeof(float));
    hipMalloc(&nn->biases2, HIDDEN_SIZE * sizeof(float));
    hipMalloc(&nn->grad_layer2, HIDDEN_SIZE * BATCH_SIZE * sizeof(float));
    init_parameters(nn->weights1, nn->biases1, HIDDEN_SIZE, HIDDEN_SIZE);
    
    hipMalloc(&nn->weights3, OUTPUT_SIZE * HIDDEN_SIZE * sizeof(float));
    hipMalloc(&nn->biases3, OUTPUT_SIZE * sizeof(float));
    hipMalloc(&nn->grad_layer3, OUTPUT_SIZE * BATCH_SIZE * sizeof(float));
    init_parameters(nn->weights3, nn->biases3, OUTPUT_SIZE, HIDDEN_SIZE);
}

void init_outputs(Outputs* op){
    hipMalloc((void**) &op->x1, HIDDEN_SIZE*BATCH_SIZE*sizeof(float));
    hipMalloc((void**) &op->a1, HIDDEN_SIZE*BATCH_SIZE*sizeof(float));

    hipMalloc((void**) &op->x2, HIDDEN_SIZE*BATCH_SIZE*sizeof(float));
    hipMalloc((void**) &op->a2, HIDDEN_SIZE*BATCH_SIZE*sizeof(float));

    hipMalloc((void**) &op->x3, OUTPUT_SIZE*BATCH_SIZE*sizeof(float));
    hipMalloc((void**) &op->a3, OUTPUT_SIZE*BATCH_SIZE*sizeof(float));

    hipMalloc((void**) &op->losses, BATCH_SIZE*sizeof(float));
}

int main(){
    NeuralNetwork nn;
    initialize_nn(&nn);

    Outputs op;
    init_outputs(&op);

}